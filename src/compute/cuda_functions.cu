#include "hip/hip_runtime.h"

#include "cuda_functions.h"
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void testKernel(int* data, int N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        data[idx] = idx * 2;
    }
}

void run_my_cuda_example()
{
    const int N = 5;
    int* d_data = nullptr;

    hipMalloc(&d_data, N * sizeof(int));
    testKernel << <1, N >> > (d_data, N);

    int h_data[N] = {};
    hipMemcpy(h_data, d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "[CUDA] Kernel result: ";
    for (int i = 0; i < N; ++i) {
        std::cout << h_data[i] << (i < N - 1 ? ", " : "\n");
    }

    hipFree(d_data);
}
